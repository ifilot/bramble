/**************************************************************************
 *                                                                        *
 *   Author: Ivo Filot <i.a.w.filot@tue.nl>                               *
 *                                                                        *
 *   BRAMBLE is free software:                                            *
 *   you can redistribute it and/or modify it under the terms of the      *
 *   GNU General Public License as published by the Free Software         *
 *   Foundation, either version 3 of the License, or (at your option)     *
 *   any later version.                                                   *
 *                                                                        *
 *   BRAMBLE is distributed in the hope that it will be useful,           *
 *   but WITHOUT ANY WARRANTY; without even the implied warranty          *
 *   of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.              *
 *   See the GNU General Public License for more details.                 *
 *                                                                        *
 *   You should have received a copy of the GNU General Public License    *
 *   along with this program.  If not, see http://www.gnu.org/licenses/.  *
 *                                                                        *
 **************************************************************************/

#include "card_manager.h"

CardManager::CardManager() {}

void CardManager::probe_cards() {
    int nDevices;

	hipGetDeviceCount(&nDevices);
	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		std::cout << "Device Number: " << i << std::endl;
		std::cout << "  Device name: " << prop.name << std::endl;
		std::cout << "  Memory Clock Rate (KHz): " << prop.memoryClockRate << std::endl;
		std::cout << "  Memory Bus Width (bits): " << prop.memoryBusWidth << std::endl;
        std::cout << "  Total memory: " << prop.totalGlobalMem << std::endl;
		std::cout << "  Peak Memory Bandwidth (GB/s): " << 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6 << std::endl << std::endl;
	}
}

int CardManager::get_num_gpus() {
    int nDevices;
    hipGetDeviceCount(&nDevices);
    return nDevices;
}

void CardManager::set_gpu_to_thread() {
    int id = omp_get_thread_num();
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, id);

    std::cout << "Setting GPU " << id << " to thread " << id << std::endl;
    hipSetDevice(id);
}

int64_t CardManager::get_memory_device(unsigned int device_id) {
    hipDeviceProp_t prop;
    if(hipGetDeviceProperties(&prop, device_id) == 0) {
        return prop.totalGlobalMem;
    } else {
        return -1;
    }
}
