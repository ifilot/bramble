#include "hip/hip_runtime.h"
/**************************************************************************
 *                                                                        *
 *   Author: Ivo Filot <i.a.w.filot@tue.nl>                               *
 *                                                                        *
 *   BRAMBLE is free software:                                            *
 *   you can redistribute it and/or modify it under the terms of the      *
 *   GNU General Public License as published by the Free Software         *
 *   Foundation, either version 3 of the License, or (at your option)     *
 *   any later version.                                                   *
 *                                                                        *
 *   BRAMBLE is distributed in the hope that it will be useful,           *
 *   but WITHOUT ANY WARRANTY; without even the implied warranty          *
 *   of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.              *
 *   See the GNU General Public License for more details.                 *
 *                                                                        *
 *   You should have received a copy of the GNU General Public License    *
 *   along with this program.  If not, see http://www.gnu.org/licenses/.  *
 *                                                                        *
 **************************************************************************/

#include "metric_analyzer_cuda.h"
#include "isorank_analyze_cuda.h"

#define gpu_err_chk(ans) { gpu_assert((ans), __FILE__, __LINE__); }

inline void gpu_assert(hipError_t code, const char *file, int line, bool abort=true) {
    if(code != hipSuccess) {
        fprintf(stderr,"GPU assert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) {
            exit(code);
        }
    }
}

MetricAnalyzerCUDA::MetricAnalyzerCUDA() {
}

/**
 * @brief      calculate distance metric using GPU
 *
 * @param[in]  cuda_device which GPU to perform this calculation on
 * @param[in]  psize       number of atom indices
 * @param[in]  start       starting permutation index
 * @param[in]  stop        stopping permutation index
 * @param[in]  _exchanges  vector holding exchanges
 * @param[in]  _mat1       distance matrix of atom 1
 * @param[in]  _mat2       distance matrix of atom 2
 * @param      _results    result vector (distance of each permutation)
 *
 * @return     lowest permutation
 */
float MetricAnalyzerCUDA::analyze_cuda(int cuda_device,
                                       size_t psize,
                                       size_t start,
                                       size_t stop,
                                       const std::vector<uint8_t>& _exchanges,
                                       const std::vector<float>& _mat1,
                                       const std::vector<float>& _mat2,
                                       std::vector<float>& _results) {
    uint8_t *exchanges;
    float *results;
    const size_t N = (stop - start);

    // specify cuda device
    gpu_err_chk(hipSetDevice(cuda_device));

    // Allocate Unified Memory – accessible from CPU or GPU
    gpu_err_chk(hipMalloc(&exchanges, N * psize * sizeof(uint8_t)));
    gpu_err_chk(hipMalloc(&results, N * sizeof(float)));

    // determine sizes
    unsigned int vecsize = _mat1.size();
    unsigned int _matsize = std::sqrt(vecsize);

    // copy to device
    gpu_err_chk(hipMemcpyToSymbol(HIP_SYMBOL(mat1), &_mat1[0], vecsize * sizeof(float)));
    gpu_err_chk(hipMemcpyToSymbol(HIP_SYMBOL(mat2), &_mat2[0], vecsize * sizeof(float)));
    gpu_err_chk(hipMemcpyToSymbol(HIP_SYMBOL(matsize), &_matsize, sizeof(unsigned int)));
    gpu_err_chk(hipMemcpyToSymbol(HIP_SYMBOL(exsize), &psize, sizeof(unsigned int)));

    // copy exchange patterns
    gpu_err_chk(hipMemcpy(exchanges, &_exchanges[start*psize], N * psize * sizeof(uint8_t), hipMemcpyHostToDevice));

    // execute kernel
    isorank_analyze_cuda<<<N, 1>>>(exchanges, results);
    gpu_err_chk(hipDeviceSynchronize());

    // copy back to host
    gpu_err_chk(hipMemcpy(&_results[0], results, N * sizeof(float), hipMemcpyDeviceToHost));

    // Free memory
    gpu_err_chk(hipFree(exchanges));
    gpu_err_chk(hipFree(results));

    return 0;
}
